#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include "solver.h"
#include "io.h"

//TODO: x is not really needed....
__global__ void reduce_kernel0(const floatType* x, const int n, floatType* x_out){
  extern __shared__ floatType sdata[];

  int tid=threadIdx.x;
  int i = blockIdx.x*blockDim.x+threadIdx.x;

  sdata[tid]= (i<n) ? x[i] :0;
  __syncthreads();

  for(unsigned int s=1; s<blockDim.x; s*=2){
    if((tid%(2*s)) == 0){
      sdata[tid]+=sdata[tid+s];
    }
  __syncthreads();
  }

  if(tid==0) x_out[blockIdx.x]= sdata[0];
}

// x_out <- x*y
// x_out needs to be reduced!!!
__global__ void reduceXY_kernel0(const floatType* x, const floatType* y, const int n, floatType* x_out){
  extern __shared__ floatType sdata[];

  int tid=threadIdx.x;
  int i = blockIdx.x*blockDim.x+threadIdx.x;

  sdata[tid]= (i<n) ? (x[i]*y[i]) :0;
  __syncthreads();

  for(unsigned int s=1; s<blockDim.x; s*=2){
    if((tid%(2*s)) == 0){
      sdata[tid]+=sdata[tid+s];
    }
  __syncthreads();
  }

  if(tid==0) x_out[blockIdx.x]= sdata[0];
}

// ab <- a' * b
void vectorDot(const floatType* a_d, const floatType* b_d, const int n, floatType* ab){
	floatType* xOut_d;
	int threads;
  int blocks;
  int s;
  size_t smem;

	getKernelConfig(n, &blocks, &threads, &smem);

	//TODO: Is it better to to allocate the memory before this point?
	//Reasons: 1.confusing interface(device/host pointers)
	//2. do this only one time not Iter times
	CUCHK(hipMalloc((void**) &xOut_d, blocks*sizeof(floatType)));

  reduceXY_kernel0<<<blocks,threads,smem>>>(a_d, b_d, n, xOut_d);
	CUCHK(hipGetLastError());

  s=blocks;
  while(s>1){
    getKernelConfig(s, &blocks, &threads, &smem);
    reduce_kernel0<<<s,threads,smem>>>(xOut_d, s, xOut_d);
		CUCHK(hipGetLastError());
    s= (s+threads-1)/(threads);
  }

	hipMemcpy(ab, xOut_d, sizeof(floatType),hipMemcpyDeviceToHost); //TODO: really copy here?

	CUCHK(hipFree(xOut_d));
}

// y <- ax + y
__global__ void axpy(const floatType a, const floatType* x, const int n, floatType* y){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<n){
		y[i]=a*x[i]+y[i];
	}
}

// y <- x + ay
__global__ void xpay(const floatType* x, const floatType a, const int n, floatType* y){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<n){
		y[i]=x[i]+a*y[i];
	}
}

//z <- ax + y
__global__ void axpyz(const floatType a, const floatType* x, const floatType* y, const int n, floatType* z){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<n){
		z[i]=a*x[i]+y[i];
	}
}



// y <- A*x
__global__ void matvec(const struct MatrixCRS A, const floatType* x, floatType* y){
	int j;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//int tid = blockIdx.x;

	if(i<A.n){
		y[i]=0;
		for(j=A.ptr[i]; j<A.ptr[i+1]; j++){
			y[i]+=A.value[j]*x[A.index[j]];
		}
	}
}//TODO: This approach is not really load balanced. Is there a better one?



//nrm <- ||x||_2
void nrm2(const floatType* x_d, const int n, floatType* nrm){
	floatType* xOut_d;
	int threads;
  int blocks;
  int s;
  size_t smem;

	getKernelConfig(n, &blocks, &threads, &smem);

	//TODO: Is it better to to allocate the memory before this point?
	//Reasons: 1.confusing interface(device/host pointers)
	//2. do this only one time not Iter times
	CUCHK(hipMalloc((void**) &xOut_d, blocks*sizeof(floatType)));

  reduceXY_kernel0<<<blocks,threads,smem>>>(x_d, x_d, n, xOut_d);
	CUCHK(hipGetLastError());

  s=blocks;
  while(s>1){
    getKernelConfig(s, &blocks, &threads, &smem);
    reduce_kernel0<<<s,threads,smem>>>(xOut_d, s, xOut_d);
		CUCHK(hipGetLastError());
    s= (s+threads-1)/(threads);
  }

	hipMemcpy(nrm, xOut_d, sizeof(floatType),hipMemcpyDeviceToHost); //TODO: really copy here?
	*nrm=sqrt(*nrm);//TODO: Do this on device?

	CUCHK(hipFree(xOut_d));
}


//nrm <- ||x||_inf
void nrmInf(const floatType* x, const int n, floatType* nrm){
	int i;
	floatType temp;

	*nrm=0;
	for(i=0; i<n; i++){
		temp=fabs(x[i]);
		if(temp>(*nrm))
			(*nrm)=temp;
	}
}/*________ UNTESTED_____*/


/***************************************
 *         Conjugate Gradient          *
 ***************************************
 p(0)    = b - Ax(0)
 r(0)    = p(0)
 rho(o)    =  <r(o),r(o)>                //TODO: need to be calculated in loop for preconditioning
 ***************************************
 for k=0,1,2,...,n-1
   q(k)      = A * p(k)                   O(n^2)
   dot_pq    = <p(k),q(k)>                O(n)
1  alpha     = rho(k) / dot_pq
2  x(k+1)    = x(k) + alpha*p(k)          O(n)
3  r(k+1)    = r(k) - alpha*q(k)          O(n)
   check convergence ||r(k+1)||_2 < eps   O(n)
	 rho(k+1)  = <r(k+1), r(k+1)>           O(n)
4  beta      = rho(k+1) / rho(k)
5  p(k+1)    = r(k+1) + beta*p(k)         O(n)
***************************************/

void cg(const struct MatrixCRS* A, const floatType* b, floatType* x, struct SolverConfig* sc){
	//floatType/** r, *p,*/ *q;
	floatType alpha, beta, rho, rho_old, dot_pq, bnrm2;
	int iter,n, nnz;
	double timeMatvec_s;
	double timeMatvec=0;

	// The device pointer
	struct MatrixCRS A_d;
	floatType* p_d;
	floatType* r_d;
	floatType* q_d;
	floatType* x_d;
	floatType* b_d;

	n=A->n;
	nnz=A->nnz;
	A_d.n=n;
	A_d.nnz=nnz;

	// allocate memory
	//r=(floatType*)malloc(n*sizeof(floatType));
	//p=(floatType*)malloc(n*sizeof(floatType));
	//q=(floatType*)malloc(n*sizeof(floatType));

	// allocate device memory
	CUCHK(hipMalloc((void**)&(A_d.value), sizeof(floatType)*nnz));
	CUCHK(hipMalloc((void**)&(A_d.index), sizeof(int)*nnz));
	CUCHK(hipMalloc((void**)&(A_d.ptr), sizeof(int)*(n+1)));
	CUCHK(hipMalloc((void**)&(x_d), sizeof(floatType)*n));
	CUCHK(hipMalloc((void**)&(p_d),sizeof(floatType)*n));
	CUCHK(hipMalloc((void**)&(r_d),sizeof(floatType)*n));
	CUCHK(hipMalloc((void**)&(q_d),sizeof(floatType)*n));
	CUCHK(hipMalloc((void**)&(b_d),sizeof(floatType)*n));

	// copy device memory
	CUCHK(hipMemcpy(A_d.value, A->value,nnz*sizeof(floatType), hipMemcpyHostToDevice));
	CUCHK(hipMemcpy(A_d.index, A->index,nnz*sizeof(int), hipMemcpyHostToDevice));
	CUCHK(hipMemcpy(A_d.ptr, A->ptr,(n+1)*sizeof(int), hipMemcpyHostToDevice));
	CUCHK(hipMemcpy(x_d, x,n*sizeof(floatType), hipMemcpyHostToDevice));
	CUCHK(hipMemcpy(b_d, b,n*sizeof(floatType), hipMemcpyHostToDevice));

	// thread configuration
	dim3 dimBlock(BLOCK_DIM);//TODO: Do this in dependency of compute capability?
	dim3 dimGrid((n+dimBlock.x-1)/dimBlock.x);

	COUT2("Start %d threads in %d blocks\n", dimBlock.x, dimGrid.x);

	//p(0)    = b - Ax(0)
	timeMatvec_s=getWTime();
	matvec<<<dimGrid, dimBlock>>>(A_d,x_d,p_d);
	timeMatvec+=getWTime()-timeMatvec_s;
	xpay<<<dimGrid, dimBlock>>>(b_d, -1.0, n, p_d);

	//calculate initial residuum
	nrm2(p_d,n,&bnrm2);
	bnrm2 = 1.0 / bnrm2;
	COUT1("bnrm %e\n", bnrm2);

	//r(0)    = p(0)
	CUCHK(hipMemcpy(r_d, p_d, n*sizeof(floatType), hipMemcpyDeviceToDevice));


	//rho(0)    =  <r(0),r(0)>
	vectorDot(r_d,r_d,n,&rho);
	COUT1("rho_0=%e\n", rho);

	for(iter = 0; iter < sc->maxIter; iter++){
		//q(k)      = A * p(k)
		timeMatvec_s=getWTime();
		matvec<<<dimGrid, dimBlock>>>(A_d,p_d,q_d);
		timeMatvec+=getWTime()-timeMatvec_s;

		//dot_pq    = <p(k),q(k)>
		vectorDot(p_d, q_d, n, &dot_pq);

		//alpha     = rho(k) / dot_pq
		alpha = rho / dot_pq;

		//x(k+1)    = x(k) + alpha*p(k)
		axpy<<<dimGrid, dimBlock>>>(alpha, p_d, n, x_d);

		//r(k+1)    = r(k) - alpha*q(k)
		axpy<<<dimGrid, dimBlock>>>(-alpha, q_d, n, r_d);

	//check convergence ||r(k+1)||_2 < eps
		//printVector(r,n);
		//printVector(x,n);
		nrm2(r_d, n, &(sc->residual));
		sc->residual*=bnrm2; // I am not sure why to correct the residuum like this, but LIS does it
		COUT2("res_%d=%e\n",iter+1, sc->residual);
		if(sc->residual <= sc->tolerance)
			break;

		rho_old=rho;

		//rho(k+1)  = <r(k+1), r(k+1)>
		vectorDot(r_d,r_d,n,&rho);

		//beta      = rho(k+1) / rho(k)
		beta = rho / rho_old;

		//p(k+1)    = r(k+1) + beta*p(k)
		xpay<<<dimGrid, dimBlock>>>(r_d, beta, n, p_d);

	}

	hipMemcpy(x, x_d,n*sizeof(floatType), hipMemcpyDeviceToHost);

	sc->iter = iter;
	sc->timeMatvec = timeMatvec;

	//printf("x=");
	//printVector(x,n);
	//printf("r=");
	//printVector(r,n);


	//free(r);
	//free(p);
	//free(q);

	// clean up the device
	CUCHK(hipFree(A_d.value));
	CUCHK(hipFree(A_d.index));
	CUCHK(hipFree(A_d.ptr));
	CUCHK(hipFree(x_d));
	CUCHK(hipFree(b_d));
	CUCHK(hipFree(p_d));
	CUCHK(hipFree(r_d));
	CUCHK(hipFree(q_d));
}

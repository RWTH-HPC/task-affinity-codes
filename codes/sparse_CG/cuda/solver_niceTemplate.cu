#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include "solver.h"
#include "io.h"

// ab <- a' * b
void vectorDot(const floatType* a, const floatType* b, const int n, floatType* ab){
/*	int i;
	(*ab)=0;
	for(i=0; i<n; i++){
		*ab += a[i]*b[i];
	}
*/



}

// y <- ax + y
void axpy(const floatType a, const floatType* x, const int n, floatType* y){
	int i;
	for(i=0; i<n; i++){
		y[i]=a*x[i]+y[i];
	}
}

// y <- x + ay
__global__ void xpay(const floatType* x, const floatType a, const int n, floatType* y){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<n){
		y[i]=x[i]+a*y[i];
	}
}

//z <- ax + y
__global__ void axpyz(const floatType a, const floatType* x, const floatType* y, const int n, floatType* z){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<n){
		z[i]=a*x[i]+y[i];
	}
}



// y <- A*x
__global__ void matvec(const struct MatrixCRS A, const floatType* x, floatType* y){
	int j;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<A.n){
		y[i]=0;
		for(j=A.ptr[i]; j<A.ptr[i+1]; j++){
			y[i]+=A.value[j]*x[A.index[j]];
		}
	}
}//TODO: This approach is not really load balanced. Is there a better one?


template <NrmPhase>
__global__ void nrm2_kernel0(const floatType* x, const int n, floatType* x_out){
  extern __shared__ double sdata[];

  int tid=threadIdx.x;
  int i = blockIdx.x*blockDim.x+threadIdx.x;
	
  sdata[tid]= (i<n) ? x[i] :0;
  __syncthreads();

  for(unsigned int s=1; s<blockDim.x; s*=2){
    if((tid%(2*s)) == 0){
      sdata[tid]+=sdata[tid+s];
    }
  __syncthreads();
  }

  if(tid==0) x_out[blockIdx.x]= sdata[0];
}

// Specialization for the WarmUp phase
template <>
__global__ void nrm2_kernel0<WarmUp>(const floatType* x, const int n, floatType* x_out){
  extern __shared__ double sdata[];

  int tid=threadIdx.x;
  int i = blockIdx.x*blockDim.x+threadIdx.x;
	
  sdata[tid]= (i<n) ? (x[i]*x[i]) :0;
  __syncthreads();

  for(unsigned int s=1; s<blockDim.x; s*=2){
    if((tid%(2*s)) == 0){
      sdata[tid]+=sdata[tid+s];
    }
  __syncthreads();
  }

  if(tid==0) x_out[blockIdx.x]= sdata[0];
}

//nrm <- ||x||_2
void nrm2(const floatType* x_d, const int n, floatType* nrm){
	floatType* xOut_d;
	int threads;
  int blocks;
  int s;
  size_t smem;

	getKernelConfig(n, &blocks, &threads, &smem);

	//TODO: Is it better to to allocate the memory before this point?
	//Reasons: 1.confusing interface(device/host pointers)
	//2. do this only one time not Iter times
	CUCHK(hipMalloc((void**) &xOut_d, blocks*sizeof(floatType)));
	
  nrm2_kernel0<WarmUp><<<blocks,threads,smem>>>(x_d, n, xOut_d);
	CUCHK(hipGetLastError());

  s=blocks;
  while(s>1){
    getKernelConfig(s, &blocks, &threads, &smem);
    nrm2_kernel0<Reduce><<<s,threads,smem>>>(xOut_d, s, xOut_d);
		CUCHK(hipGetLastError());
    s= (s+threads-1)/(threads);
  }

	hipMemcpy(nrm, xOut_d, sizeof(floatType),hipMemcpyDeviceToHost); //TODO: really copy here?
	*nrm=sqrt(*nrm);

	CUCHK(hipFree(xOut_d));
}


//nrm <- ||x||_inf
void nrmInf(const floatType* x, const int n, floatType* nrm){
	int i;
	floatType temp;

	*nrm=0;
	for(i=0; i<n; i++){
		temp=fabs(x[i]);
		if(temp>(*nrm))
			(*nrm)=temp;
	}
}/*________ UNTESTED_____*/


/***************************************
 *         Conjugate Gradient          *
 ***************************************
 p(0)    = b - Ax(0)
 r(0)    = p(0)
 rho(o)    =  <r(o),r(o)>                //TODO: need to be calculated in loop for preconditioning 
 ***************************************
 for k=0,1,2,...,n-1
   q(k)      = A * p(k)                   O(n^2)
   dot_pq    = <p(k),q(k)>                O(n)
1  alpha     = rho(k) / dot_pq            
2  x(k+1)    = x(k) + alpha*p(k)          O(n)
3  r(k+1)    = r(k) - alpha*q(k)          O(n)
   check convergence ||r(k+1)||_2 < eps   O(n)
	 rho(k+1)  = <r(k+1), r(k+1)>           O(n) 
4  beta      = rho(k+1) / rho(k)
5  p(k+1)    = r(k+1) + beta*p(k)         O(n)
***************************************/

void cg(const struct MatrixCRS* A, const floatType* b, floatType* x, struct SolverConfig* sc){
	floatType/** r, *p,*/ *q;
	floatType alpha, beta, rho, rho_old, dot_pq, bnrm2;
	int iter,n, nnz;

	// The device pointer
	struct MatrixCRS A_d;
	floatType* p_d;
	floatType* r_d;
	floatType* x_d;
	floatType* b_d;
	
	n=A->n;
	nnz=A->nnz;
	A_d.n=n;
	A_d.nnz=nnz;	

	// allocate memory
	//r=(floatType*)malloc(n*sizeof(floatType));
	//p=(floatType*)malloc(n*sizeof(floatType));
	q=(floatType*)malloc(n*sizeof(floatType));

	// allocate device memory
	CUCHK(hipMalloc((void**)&(A_d.value), sizeof(floatType)*nnz));
	CUCHK(hipMalloc((void**)&(A_d.index), sizeof(int)*nnz));
	CUCHK(hipMalloc((void**)&(A_d.ptr), sizeof(int)*(n+1)));
	CUCHK(hipMalloc((void**)&(x_d), sizeof(floatType)*n));
	CUCHK(hipMalloc((void**)&(p_d),sizeof(floatType)*n));
	CUCHK(hipMalloc((void**)&(r_d),sizeof(floatType)*n));
	CUCHK(hipMalloc((void**)&(b_d),sizeof(floatType)*n));

	// copy device memory
	CUCHK(hipMemcpy(A_d.value, A->value,nnz*sizeof(floatType), hipMemcpyHostToDevice));
	CUCHK(hipMemcpy(A_d.index, A->index,nnz*sizeof(int), hipMemcpyHostToDevice));
	CUCHK(hipMemcpy(A_d.ptr, A->ptr,(n+1)*sizeof(int), hipMemcpyHostToDevice));
	CUCHK(hipMemcpy(x_d, x,n*sizeof(floatType), hipMemcpyHostToDevice));
	CUCHK(hipMemcpy(b_d, b,n*sizeof(floatType), hipMemcpyHostToDevice));

	// thread configuration 
	dim3 dimBlock(BLOCK_DIM);//TODO: Do this in dependency of compute capability?
	dim3 dimGrid((n+dimBlock.x-1)/dimBlock.x);
	COUT2("Start %d threads in %d blocks\n", dimBlock.x, dimGrid.x);
	
	printf("SOLVING CG\n");
	printf("NNZ      : %8d\n", nnz);
	printf("N        : %8d\n", n);
	printf("MAX_ITER : %8d\n", sc->maxIter);
	printf("TOLERANCE: %8.0e\n", sc->tolerance);
	
	//p(0)    = b - Ax(0)
	matvec<<<dimGrid, dimBlock>>>(A_d,x_d,p_d);
	xpay<<<dimGrid, dimBlock>>>(b_d, -1.0, n, p_d);
	
	//calculate initial residuum
	nrm2(p_d,n,&bnrm2);
	bnrm2 = 1.0 / bnrm2;
	COUT1("bnrm %e\n", bnrm2);

	//r(0)    = p(0)
	CUCHK(hipMemcpy(r_d, p_d, n*sizeof(floatType), CudaMemcpyDeviceToDevice));	


	//rho(0)    =  <r(0),r(0)>
	vectorDot(r,r,n,&rho);

	/*for(iter = 0; iter < sc->maxIter; iter++){
		//q(k)      = A * p(k)
		//matvec(A,p,q);

		//dot_pq    = <p(k),q(k)>
		vectorDot(p, q, n, &dot_pq);

		//alpha     = rho(k) / dot_pq
		alpha = rho / dot_pq;

		//x(k+1)    = x(k) + alpha*p(k)
		axpy(alpha, p, n, x); 

		//r(k+1)    = r(k) - alpha*q(k)
		axpy(-alpha, q, n, r);

   	//check convergence ||r(k+1)||_2 < eps
		//printVector(r,n);
		//printVector(x,n);
		nrm2(r, n, &(sc->residual));
		sc->residual*=bnrm2; // I am not sure why to correct the residuum like this, but LIS does it 
		COUT2("res_%d=%e\n",iter+1, sc->residual);
		if(sc->residual <= sc->tolerance)
			break;

		rho_old=rho;

		//rho(k+1)  = <r(k+1), r(k+1)>
		vectorDot(r,r,n,&rho);
	
		//beta      = rho(k+1) / rho(k)
		beta = rho / rho_old;

		//p(k+1)    = r(k+1) + beta*p(k)
		xpay(r, beta, n, p);

	}	*/

	hipMemcpy(x, p_d,n*sizeof(floatType), hipMemcpyDeviceToHost);//TODO: CORRECT THIS TO X

	printf("RESIDUAL : %8.0e\n", sc->residual);
	printf("ITER     : %8d\n", iter);

	//printf("x=");	
	//printVector(x,n);	
	//printf("r=");	
	//printVector(r,n);	


	//free(r);
	//free(p);
	free(q);

	// clean up the device
	CUCHK(hipFree(A_d.value));
	CUCHK(hipFree(A_d.index));
	CUCHK(hipFree(A_d.ptr));
	CUCHK(hipFree(x_d));
	CUCHK(hipFree(b_d));
	CUCHK(hipFree(p_d));
	CUCHK(hipFree(r_d));
}

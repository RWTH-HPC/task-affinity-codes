#include "hip/hip_runtime.h"
#include "errorcheck.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// Check if the normalized residual (which is a ration of the final and the
// inital residual) is smaller than the specified CG tolerance. 0 is returned
// in case of errors, 1 otherwise.
int
check_error(const floatType bnrm2, const floatType residual,
    const floatType cg_tol)
{
	return ((residual / bnrm2) <= cg_tol ? 1 : 0);
}

// Returns the maximum absolute error
floatType
get_max_error(const floatType *const x, const int n)
{
	floatType maxError = 0.0;
	int i;

	for (i = 0; i < n; i++) {
		// FIXME: This will only work if solution is 1
		floatType err = fabs(x[i] - 1);
		if (err > maxError)
			maxError = err;
	}

	return maxError;
}

// Calculate the current residual for error checking. You must not change this
// function, it is not used to during the algorithm. There is no need to
// parallelize it.
floatType
get_residual(const struct MatrixCRS *const A, const floatType *const b,
    const floatType *const x)
{
	int i, j;
	floatType *y;
	floatType residual;

	// Allocate residual vector
	y = (floatType*)malloc(A->n * sizeof(floatType));

	// y = A * x
	for (i = 0; i < A->n; i++) {
		y[i] = 0;

		for (j = A->ptr[i]; j < A->ptr[i + 1]; j++)
			y[i] += A->value[j] * x[A->index[j]];
	}

	// y = | b - y |
	for (i = 0; i < A->n; i++)
		y[i] = fabs(b[i]-y[i]);

	// residual = || y ||_2
	residual = 0;
	for (i = 0; i < A->n; i++)
		residual += y[i] * y[i];
	residual = sqrt(residual);

	// Clean up
	free(y);

	return residual;
}
